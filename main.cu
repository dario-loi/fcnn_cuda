
#include "tensor.hpp"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <numeric>
#include <span>

int main()
{

    auto T = TensorF32({ 1, 2, 2 * 2 });
    auto T2 = TensorF32({ 1, 2, 2 * 2 });

    T.randn();
    T2.randn(0, 5);

    auto Z = (T * T2);
    auto Z2 = (T + T2);
    auto Z3 = (Z * Z2);

    std::cout << sizeof(TensorF32) << std::endl;
    std::cout << sizeof(TensorF64) << std::endl;
    Z3.backward();

    // Z.print();
    Z.print_info();
    Z.creator_op->print_info();
    Z.print_grad();
    T.print_grad();
    T2.print_grad();

    T.print();
    T2.print();

    return 0;
}
