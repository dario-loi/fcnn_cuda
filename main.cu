#include "aliases.hpp"
#include "backend_cpu.hpp"
#include "tensor.hpp"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <numeric>
#include <span>
int main()
{
    CPUTensorF32 T = CPUTensorF32 { { 2, 2, 2, 4 } };

    printf("Size of T: %d\n", T.size);
    printf("Rank of T: %d\n", T.ndim);
    T.fill(1.0f);

    T.rand();
    T.pretty_print();
    T.print();
    return 0;
}
